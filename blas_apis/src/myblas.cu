
#include "myblas.h"

// ================ LEVEL 1 APIS ================
template <int NB, int NT>
void myblas_api_SAXPY(int n, const float *alpha, const float *x, float *y) {
    int size = n * sizeof(float);
    // Allocate memory to GPU
    float *d_x;
    hipMalloc(&d_x, size);
    float *d_y;
    hipMalloc(&d_y, size);
    float *d_alpha;
    hipMalloc(&d_alpha, sizeof(float));

    // Copy memory to GPU
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, alpha, sizeof(float), hipMemcpyHostToDevice);

    // Initialize kernel and run function
    int threadsPerBlock = n;
    int numBlocks = 1;
    myblas_SAXPY<<<NB, NT>>>(n, d_alpha, d_x, 1, d_y, 1);

    // Copy memory back
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_alpha);
}

template <int NB, int NT>
void myblas_api_SDOT(int n, const float *x, const float *y, float *result) {
    int size = n * sizeof(float);
    // Allocate memory to GPU
    float *d_x;
    hipMalloc(&d_x, size);
    float *d_y;
    hipMalloc(&d_y, size);
    float *d_result;
    hipMalloc(&d_result, sizeof(float));

    // Copy memory to GPU
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    // Initialize kernel and run function
    int threadsPerBlock = n;
    int numBlocks = 1;
    myblas_SDOT<<<NB, NT>>>(n, d_x, 1, d_y, 1, d_result);

    // Copy memory back
    hipMemcpy(result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);
}

// ================ LEVEL 2 APIS ================
template <int NB, int NT>
void myblas_api_SGEMV(int m, int n, const float *alpha, const float *A,
                      const float *x, const float *beta, float *y) {
    int size_v = m * sizeof(float);
    int size_m = n * m * sizeof(float);

    // Allocate memory to gpu
    float *d_A;
    hipMalloc(&d_A, size_m);
    float *d_x;
    hipMalloc(&d_x, size_v);
    float *d_y;
    hipMalloc(&d_y, size_v);
    float *d_alpha;
    hipMalloc(&d_alpha, sizeof(float));
    float *d_beta;
    hipMalloc(&d_beta, sizeof(float));

    // Copy memory to GPU
    hipMemcpy(d_A, A, size_m, hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, size_v, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size_v, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, alpha, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_beta, beta, sizeof(float), hipMemcpyHostToDevice);

    // Run Kernel
    myblas_SGEMV<<<NB, NT>>>('N', m, n, d_alpha, d_A, 1, d_x, 1, d_beta, d_y,
                             1);
    // Copy memory back from GPU
    hipMemcpy(y, d_y, size_v, hipMemcpyDeviceToHost);

    // Free memory in GPU
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_alpha);
    hipFree(d_beta);
}

// Assumed to be upper triangular and not unit triangular
template <int NB, int NT>
void myblas_api_STRSV(int n, const float *A, float *x) {
    int size_v = n * sizeof(float);
    int size_m = n * n * sizeof(float);

    // Allocate memory in GPU
    float *d_A;
    hipMalloc(&d_A, size_m);
    float *d_x;
    hipMalloc(&d_x, size_v);

    // Copy memory to GPU
    hipMemcpy(d_A, A, size_m, hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, size_v, hipMemcpyHostToDevice);

    // Execute kernel
    myblas_STRSV<<<NB, NT>>>('U', 'N', 'N', n, d_A, 1, d_x, 1);

    // Free memory in GPU
    hipFree(d_A);
    hipFree(d_x);
}

// ================ LEVEL 3 APIS ================
template <int NB, int NT>
void myblas_SGEMM(int m, int n, int k, const float *alpha, const float *A,
                  const float *B, const float *beta, float *C) {
    int size_A = m * k * sizeof(float);
    int size_B = n * k * sizeof(float);
    int size_C = m * n * sizeof(float);
    // Allocate memory in GPU
    float *d_A;
    hipMalloc(&d_A, size_A);
    float *d_B;
    hipMalloc(&d_B, size_B);
    float *d_C;
    hipMalloc(&d_C, size_C);
    float *d_alpha;
    hipMalloc(&d_alpha, sizeof(float));
    float *d_beta;
    hipMalloc(&d_beta, sizeof(float));
    // Copy memory to GPU
    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, size_C, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, alpha, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_beta, beta, sizeof(float), hipMemcpyHostToDevice);

    // Run kernel
    myblas_SGEMM<<<NB, NT>>>('N', 'N', n, m, k, d_alpha, d_A, 1, d_B, 1, d_beta,
                             d_C, 1);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_alpha);
    hipFree(d_beta);
}

// Assumed to be upper triangular and not unit triangular
template <int NB, int NT>
void myblas_api_STRSM(int m, int n, const float *alpha, const float *A,
                      float *B) {
    int size = n * m * sizeof(float);
    // Allocate memory in GPU
    float *d_A;
    hipMalloc(&d_A, size);
    float *d_B;
    hipMalloc(&d_B, size);
    float *d_alpha;
    hipMalloc(&d_alpha, sizeof(float));

    // Copy memory to GPU
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, alpha, sizeof(float), hipMemcpyHostToDevice);

    // Run kernel
    myblas_STRSM<<<NB, NT>>>('L', 'L', 'N', 'N', n, m, d_alpha, d_A, 1, d_B, 1);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_alpha);
}