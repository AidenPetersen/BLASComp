
#include <stdio.h>

#include "myblas.h"

#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        // if (abort) exit(code);
    }
}

// ================ LEVEL 1 APIS ================
void myblas_api_SAXPY(int n, const float *alpha, const float *x, float *y) {
    int size = n * sizeof(float);
    // Allocate memory to GPU
    float *d_x;
    hipMalloc(&d_x, size);
    float *d_y;
    hipMalloc(&d_y, size);
    float *d_alpha;
    hipMalloc(&d_alpha, sizeof(float));

    // Copy memory to GPU
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, alpha, sizeof(float), hipMemcpyHostToDevice);

    // Initialize kernel and run function
    myblas_SAXPY<<<n / 128, 128>>>(n, d_alpha, d_x, 1, d_y, 1);

    // Copy memory back
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_alpha);
}

void myblas_api_SDOT(int n, const float *x, const float *y, float *result) {
    int size = n * sizeof(float);
    // Allocate memory to GPU
    float *d_x;
    hipMalloc(&d_x, size);
    float *d_y;
    hipMalloc(&d_y, size);
    float *d_result;
    hipMalloc(&d_result, sizeof(float));

    // Copy memory to GPU
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    // Initialize kernel and run function
    myblas_SDOT<<<n / 128, 128>>>(n, d_x, 1, d_y, 1, d_result);

    // Copy memory back
    hipMemcpy(result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);
}

// ================ LEVEL 2 APIS ================
void myblas_api_SGEMV(int m, int n, const float *alpha, const float *A,
                      const float *x, const float *beta, float *y) {
    int size_v = m * sizeof(float);
    int size_m = n * m * sizeof(float);

    // create memory stream

    // Allocate memory to gpu
    float *d_A;
    hipMalloc(&d_A, size_m);
    float *d_x;
    hipMalloc(&d_x, size_v);
    float *d_y;
    hipMalloc(&d_y, size_v);
    float *d_alpha;
    hipMalloc(&d_alpha, sizeof(float));
    float *d_beta;
    hipMalloc(&d_beta, sizeof(float));

    // Copy memory to GPU
    hipMemcpy(d_A, A, size_m, hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, size_v, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size_v, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, alpha, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_beta, beta, sizeof(float), hipMemcpyHostToDevice);

    // Run Kernel
    myblas_SGEMV<<<n / 128, 128>>>('N', m, n, d_alpha, d_A, 1, d_x, 1, d_beta,
                                   d_y, 1);
    // Copy memory back from GPU
    hipMemcpy(y, d_y, size_v, hipMemcpyDeviceToHost);

    // Free memory in GPU
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_alpha);
    hipFree(d_beta);
}

// Assumed to be upper triangular and not unit triangular
void myblas_api_STRSV(int n, const float *A, float *x) {
    int size_v = n * sizeof(float);
    int size_m = n * n * sizeof(float);

    // Allocate memory in GPU
    float *d_A;
    hipMalloc(&d_A, size_m);
    float *d_x;
    hipMalloc(&d_x, size_v);

    // Copy memory to GPU
    hipMemcpy(d_A, A, size_m, hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, size_v, hipMemcpyHostToDevice);

    // Execute kernel
    myblas_STRSV<<<n / 128, 128>>>('U', 'N', 'N', n, d_A, 1, d_x, 1);

    // Free memory in GPU
    hipFree(d_A);
    hipFree(d_x);
}

// ================ LEVEL 3 APIS ================
void myblas_api_SGEMM(int m, int n, int k, const float *alpha, const float *A,
                      const float *B, const float *beta, float *C) {
    int size_A = m * k * sizeof(float);
    int size_B = n * k * sizeof(float);
    int size_C = m * n * sizeof(float);
    // Allocate memory in GPU
    float *d_A;
    hipMalloc(&d_A, size_A);
    float *d_B;
    hipMalloc(&d_B, size_B);
    float *d_C;
    hipMalloc(&d_C, size_C);
    float *d_alpha;
    hipMalloc(&d_alpha, sizeof(float));
    float *d_beta;
    hipMalloc(&d_beta, sizeof(float));
    // Copy memory to GPU
    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, size_C, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, alpha, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_beta, beta, sizeof(float), hipMemcpyHostToDevice);

    // Run kernel
    dim3 threadsPerBlock(32, 32);
    myblas_SGEMM<<<n / 32, threadsPerBlock>>>('N', 'N', m, n, k, d_alpha, d_A,
                                              1, d_B, 1, d_beta, d_C, 1);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_alpha);
    hipFree(d_beta);
}

// Assumed to be upper triangular and not unit triangular
void myblas_api_STRSM(int m, int n, const float *alpha, const float *A,
                      float *B) {
    int size = n * m * sizeof(float);
    // Allocate memory in GPU
    float *d_A;
    hipMalloc(&d_A, size);
    float *d_B;
    hipMalloc(&d_B, size);
    float *d_alpha;
    hipMalloc(&d_alpha, sizeof(float));

    // Copy memory to GPU
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, alpha, sizeof(float), hipMemcpyHostToDevice);

    // Run kernel
    dim3 threadsPerBlock(n, n);
    myblas_STRSM<<<n / 128, threadsPerBlock>>>('L', 'L', 'N', 'N', m, n,
                                               d_alpha, d_A, 1, d_B, 1);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_alpha);
}