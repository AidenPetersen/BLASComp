#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "hipblas.h"
// ================ LEVEL 1 APIS ================
void cublas_api_SAXPY(int n, const float *alpha, const float *x, float *y) {
    hipblasHandle_t cublasH = NULL;

    hipblasCreate(&cublasH);

    // Run function
    hipblasSaxpy(cublasH, n, alpha, x, 1, y, 1);

    hipblasDestroy(cublasH);
}

void cublas_api_SDOT(int n, const float *x, const float *y, float *result) {
    hipblasHandle_t cublasH = NULL;
    hipblasCreate(&cublasH);

    // Run function
    hipblasSdot(cublasH, n, x, 1, y, 1, result);

    hipblasDestroy(cublasH);
}

// ================ LEVEL 2 APIS ================
void cublas_api_SGEMV(int m, int n, const float *alpha, const float *A,
                      const float *x, const float *beta, float *y) {
    hipblasHandle_t cublasH = NULL;
    hipblasCreate(&cublasH);

    // Run function
    hipblasSgemv(cublasH, HIPBLAS_OP_N, m, n, alpha, A, m, x, 1, beta, y, 1);

    hipblasDestroy(cublasH);
}

// Assumed to be upper triangular and not unit triangular
void cublas_api_STRSV(int n, const float *A, float *x) {
    hipblasHandle_t cublasH = NULL;
    hipblasCreate(&cublasH);

    // Run function
    hipblasStrsv(cublasH, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N,
                HIPBLAS_DIAG_NON_UNIT, n, A, n, x, 1);

    hipblasDestroy(cublasH);
}

// ================ LEVEL 3 APIS ================
void cublas_api_SGEMM(int m, int n, int k, const float *alpha, const float *A,
                      const float *B, const float *beta, float *C) {
    hipblasHandle_t cublasH = NULL;
    hipblasCreate(&cublasH);

    // Run kernel
    hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, alpha, A, m, B, k,
                beta, C, m);

    hipblasDestroy(cublasH);
}

// Assumed to be upper triangular and not unit triangular
void cublas_api_STRSM(int m, int n, const float *alpha, const float *A,
                      float *B) {
    hipblasHandle_t cublasH = NULL;
    hipblasCreate(&cublasH);

    // Run function
    hipblasStrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N,
                HIPBLAS_DIAG_NON_UNIT, m, n, alpha, A, m, B, m);

    hipblasDestroy(cublasH);
}