#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "hipblas.h"
// ================ LEVEL 1 APIS ================
void cublas_api_SAXPY(int n, const float *alpha, const float *x, float *y) {
    hipblasHandle_t cublasH = NULL;
    hipblasCreate(&cublasH);

    int size = n * sizeof(float);
    // Allocate memory to GPU
    float *d_x;
    hipMalloc(&d_x, size);
    float *d_y;
    hipMalloc(&d_y, size);
    float *d_alpha;
    hipMalloc(&d_alpha, sizeof(float));

    // Copy memory to GPU
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, alpha, sizeof(float), hipMemcpyHostToDevice);

    // run
    hipblasSaxpy(cublasH, n, d_alpha, d_x, 1, d_y, 1);

    // Copy memory back
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_alpha);

    hipblasDestroy(cublasH);
}

void cublas_api_SDOT(int n, const float *x, const float *y, float *result) {
    hipblasHandle_t cublasH = NULL;
    hipblasCreate(&cublasH);

    int size = n * sizeof(float);
    // Allocate memory to GPU
    float *d_x;
    hipMalloc(&d_x, size);
    float *d_y;
    hipMalloc(&d_y, size);
    float *d_result;
    hipMalloc(&d_result, sizeof(float));

    // Copy memory to GPU
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    // Initialize kernel and run function
    int threadsPerBlock = n;
    int numBlocks = 1;
    hipblasSdot(cublasH, n, d_x, 1, d_y, 1, d_result);

    // Copy memory back
    hipMemcpy(result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);
    hipblasDestroy(cublasH);
}

// ================ LEVEL 2 APIS ================
void cublas_api_SGEMV(int m, int n, const float *alpha, const float *A,
                      const float *x, const float *beta, float *y) {
    hipblasHandle_t cublasH = NULL;
    hipblasCreate(&cublasH);
    int size_v = m * sizeof(float);
    int size_m = n * m * sizeof(float);

    // Allocate memory to gpu
    float *d_A;
    hipMalloc(&d_A, size_m);
    float *d_x;
    hipMalloc(&d_x, size_v);
    float *d_y;
    hipMalloc(&d_y, size_v);
    float *d_alpha;
    hipMalloc(&d_alpha, sizeof(float));
    float *d_beta;
    hipMalloc(&d_beta, sizeof(float));

    // Copy memory to GPU
    hipMemcpy(d_A, A, size_m, hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, size_v, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size_v, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, alpha, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_beta, beta, sizeof(float), hipMemcpyHostToDevice);

    // Run Kernel
    hipblasSgemv(cublasH, HIPBLAS_OP_N, m, n, d_alpha, d_A, 1, d_x, 1, d_beta,
                d_y, 1);
    // Copy memory back from GPU
    hipMemcpy(y, d_y, size_v, hipMemcpyDeviceToHost);

    // Free memory in GPU
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_alpha);
    hipFree(d_beta);
    hipblasDestroy(cublasH);
}

// Assumed to be upper triangular and not unit triangular
void cublas_api_STRSV(int n, const float *A, float *x) {
    hipblasHandle_t cublasH = NULL;
    hipblasCreate(&cublasH);
    int size_v = n * sizeof(float);
    int size_m = n * n * sizeof(float);

    // Allocate memory in GPU
    float *d_A;
    hipMalloc(&d_A, size_m);
    float *d_x;
    hipMalloc(&d_x, size_v);

    // Copy memory to GPU
    hipMemcpy(d_A, A, size_m, hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, size_v, hipMemcpyHostToDevice);

    // Execute kernel
    hipblasStrsv(HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n,
                d_A, 1, d_x, 1);

    // Free memory in GPU
    hipFree(d_A);
    hipFree(d_x);
    hipblasDestroy(cublasH);
}

// ================ LEVEL 3 APIS ================
void cublas_api_SGEMM(int m, int n, int k, const float *alpha, const float *A,
                      const float *B, const float *beta, float *C) {
    hipblasHandle_t cublasH = NULL;
    hipblasCreate(&cublasH);
    int size_A = m * k * sizeof(float);
    int size_B = n * k * sizeof(float);
    int size_C = m * n * sizeof(float);
    // Allocate memory in GPU
    float *d_A;
    hipMalloc(&d_A, size_A);
    float *d_B;
    hipMalloc(&d_B, size_B);
    float *d_C;
    hipMalloc(&d_C, size_C);
    float *d_alpha;
    hipMalloc(&d_alpha, sizeof(float));
    float *d_beta;
    hipMalloc(&d_beta, sizeof(float));
    // Copy memory to GPU
    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, size_C, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, alpha, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_beta, beta, sizeof(float), hipMemcpyHostToDevice);

    // Run kernel
    hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, d_alpha, d_A, m,
                d_B, k, d_beta, d_C, m);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_alpha);
    hipFree(d_beta);
    hipblasDestroy(cublasH);
}

// Assumed to be upper triangular and not unit triangular
void cublas_api_STRSM(int m, int n, const float *alpha, const float *A,
                      float *B) {
    hipblasHandle_t cublasH = NULL;
    hipblasCreate(&cublasH);
    int size = n * m * sizeof(float);
    // Allocate memory in GPU
    float *d_A;
    hipMalloc(&d_A, size);
    float *d_B;
    hipMalloc(&d_B, size);
    float *d_alpha;
    hipMalloc(&d_alpha, sizeof(float));

    // Copy memory to GPU
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, alpha, sizeof(float), hipMemcpyHostToDevice);

    // Run kernel
    hipblasStrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N,
                HIPBLAS_DIAG_NON_UNIT, m, n, d_alpha, d_A, m, d_B, m);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_alpha);
    hipblasDestroy(cublasH);
}