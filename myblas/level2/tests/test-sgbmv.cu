#include <stdio.h>

#include "level2/level2.h"

int main() {
    int N = 6;
    size_t size = N * sizeof(float);

    // Allocate h_A h_B in host memory
    static const float h_A[36] = {1, 1, 0, 0, 0, 0, 1, 2, 1, 0, 0, 0,
                                  0, 1, 3, 1, 0, 0, 0, 0, 1, 4, 1, 0,
                                  0, 0, 0, 1, 5, 1, 0, 0, 0, 0, 1, 6};
    float* h_x = (float*)malloc(size);
    float* h_y = (float*)malloc(size);
    float* h_alpha = (float*)malloc(sizeof(float));
    float* h_beta = (float*)malloc(sizeof(float));

    // Initialize inputs
    *h_alpha = 1;
    *h_beta = 1;
    for (int i = 0; i < N; i++) {
        h_x[i] = i + 1;
        h_y[i] = 0;
    }
    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size * N);
    float* d_x;
    hipMalloc(&d_x, size);
    float* d_y;
    hipMalloc(&d_y, size);
    float* d_alpha;
    hipMalloc(&d_alpha, sizeof(float));
    float* d_beta;
    hipMalloc(&d_beta, sizeof(float));

    // Copy vectors from host to device
    hipMemcpy(d_A, h_A, N * size, hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, h_alpha, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_beta, h_beta, sizeof(float), hipMemcpyHostToDevice);

    myblas_SGBMV<<<1, N>>>('N', N, N, 1, 1, d_alpha, d_A, 1, d_x, 1, d_beta,
                           d_y, 1);

    hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        printf("%f\n", h_y[i]);
    }
    exit(!(h_y[0] == 3 && h_y[1] == 8 && h_y[2] == 15 && h_y[3] == 24 &&
           h_y[4] == 35 && h_y[5] == 41));
}