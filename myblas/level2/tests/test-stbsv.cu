#include "hip/hip_runtime.h"
#include <stdio.h>

#include "level2/level2.h"

int main() {
    int N = 4;
    size_t size = N * sizeof(float);

    // Allocate h_A h_B in host memory
    static const float h_A[16] = {2, 0, 0, 0, 1, 5, 0, 0,
                                  1, 2, 3, 0, 1, 2, 3, 4};
    float* h_x = (float*)malloc(size);
    float* h_y = (float*)malloc(size);
    float* h_alpha = (float*)malloc(sizeof(float));
    float* h_beta = (float*)malloc(sizeof(float));

    // Initialize inputs
    *h_alpha = 1;
    *h_beta = 1;
    for (int i = 0; i < N; i++) {
        h_x[i] = i + 1;
        h_y[i] = 0;
    }
    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size * N);
    float* d_x;
    hipMalloc(&d_x, size);

    // Copy vectors from host to device
    hipMemcpy(d_A, h_A, N * size, hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);

    myblas_STBSV<<<1, N>>>('L', 'N', 'N', N, 0, d_A, 1, d_x, 1);

    hipMemcpy(h_x, d_x, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        printf("%f\n", h_x[i]);
    }
    exit(!(abs(h_x[0] - (1.0 / 2.0)) < 0.001 &&
           abs(h_x[1] - (3.0 / 10.0)) < 0.001 &&
           abs(h_x[2] - (19.0 / 30.0)) < 0.001 &&
           abs(h_x[3] - (1.0 / 4.0)) < 0.001));
}