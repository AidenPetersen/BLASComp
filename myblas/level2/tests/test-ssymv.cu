#include <stdio.h>

#include "level2/level2.h"

int main() {
    int N = 4;
    size_t size = N * sizeof(float);

    // Allocate h_A h_B in host memory
    static const float h_A[16] = {1, 2, 3, 4, 2, 1, 2, 3,
                                  3, 2, 1, 2, 4, 3, 2, 1};
    float* h_x = (float*)malloc(size);
    float* h_y = (float*)malloc(size);
    float* h_alpha = (float*)malloc(sizeof(float));
    float* h_beta = (float*)malloc(sizeof(float));

    // Initialize inputs
    *h_alpha = 1;
    *h_beta = 1;
    for (int i = 0; i < N; i++) {
        h_x[i] = i + 1;
        h_y[i] = 0;
    }
    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size * N);
    float* d_x;
    hipMalloc(&d_x, size);
    float* d_y;
    hipMalloc(&d_y, size);
    float* d_alpha;
    hipMalloc(&d_alpha, sizeof(float));
    float* d_beta;
    hipMalloc(&d_beta, sizeof(float));

    // Copy vectors from host to device
    hipMemcpy(d_A, h_A, N * size, hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, h_alpha, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_beta, h_beta, sizeof(float), hipMemcpyHostToDevice);

    myblas_SSYMV<<<1, N>>>('L', N, d_alpha, d_A, 1, d_x, 1, d_beta, d_y, 1);

    hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        printf("%f\n", h_y[i]);
    }
    exit(!(h_y[0] == 30 && h_y[1] == 22 && h_y[2] == 18 && h_y[3] == 20));
}