#include "hip/hip_runtime.h"
#include <stdio.h>

#include "level2/level2.h"

int main() {
    int N = 4;
    size_t size = N * sizeof(float);

    // Allocate h_A h_B in host memory
    static float h_A[16] = {1, 2,  3,  4,  5,  6,  7,  8,
                            9, 10, 11, 12, 13, 14, 15, 16};
    float* h_x = (float*)malloc(size);
    float* h_y = (float*)malloc(size);
    float* h_alpha = (float*)malloc(sizeof(float));

    // Initialize inputs
    *h_alpha = 1;
    for (int i = 0; i < N; i++) {
        h_x[i] = 1;
        h_y[i] = 1;
    }
    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size * N);
    float* d_x;
    hipMalloc(&d_x, size);
    float* d_y;
    hipMalloc(&d_y, size);
    float* d_alpha;
    hipMalloc(&d_alpha, sizeof(float));

    // Copy vectors from host to device
    hipMemcpy(d_A, h_A, N * size, hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, h_alpha, sizeof(float), hipMemcpyHostToDevice);
    int numBlocks = 1;
    dim3 threadsPerBlock(N, N);
    myblas_SGER<<<numBlocks, threadsPerBlock>>>(N, N, d_alpha, d_x, 1, d_y, 1,
                                                d_A, 1);

    hipMemcpy(h_A, d_A, size * N, hipMemcpyDeviceToHost);
    int result = 0;
    for (int i = 0; i < N * N; i++) {
        if (abs(((float)i + 2.0) - h_A[i]) > 0.01) {
            result = 1;
        }
    }
    exit(result);
}