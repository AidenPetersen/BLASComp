#include "level1/level1.h"
#include <stdio.h>

int main(){

    // Allocate h_A h_B in host memory
    float h_A = 0.0;
    float h_B = 2.0;
    float h_C;
    float h_S;


    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, 1);
    float* d_B;
    hipMalloc(&d_B, 1);
    float* d_C;
    hipMalloc(&d_C, 1);    
    float* d_S;
    hipMalloc(&d_S, 1);

    // Copy vectors from host to device
    hipMemcpy(d_A, &h_A, 1, hipMemcpyHostToDevice);
    hipMemcpy(d_B, &h_B, 1, hipMemcpyHostToDevice);

    myblas_SROTG<<<1, 1>>>(d_A, d_B, d_C, d_S);
    hipMemcpy(&h_A, d_A, 1, hipMemcpyDeviceToHost);
    hipMemcpy(&h_B, d_B, 1, hipMemcpyDeviceToHost);
    hipMemcpy(&h_C, d_C, 1, hipMemcpyDeviceToHost);
    hipMemcpy(&h_S, d_S, 1, hipMemcpyDeviceToHost);

    exit(h_A == 2.0 && h_B == 1.0 && h_C == 0.0 && h_S == 1.0);
}