#include <stdio.h>

#include "level1/level1.h"

int main() {
    int N = 5;
    size_t size = N * sizeof(float);

    // Allocate h_x h_alpha in host memory
    float *h_x = (float *)malloc(size);
    float *h_y = (float *)malloc(size);
    float *h_alpha = (float *)malloc(sizeof(float));

    // Initialize inputs
    *h_alpha = 3;
    for (int i = 0; i < 5; i++) {
        h_x[i] = i;
    }

    for (int i = 0; i < 5; i++) {
        h_y[i] = 1.0;
    }

    // Allocate vectors in device memory
    float *d_x;
    hipMalloc(&d_x, size);
    float *d_y;
    hipMalloc(&d_y, size);
    float *d_alpha;
    hipMalloc(&d_alpha, sizeof(float));

    // Copy vectors from host to device
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);

    hipMemcpy(d_alpha, h_alpha, sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    myblas_SAXPY<<<blocksPerGrid, threadsPerBlock>>>(N, d_alpha, d_x, 1, d_y,
                                                     1);

    hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 5; i++) {
        if (h_y[i] != i * (*h_alpha) + 1) {
            exit(1);
        }
    }
    exit(0);
}